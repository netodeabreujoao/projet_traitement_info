#include "hip/hip_runtime.h"
#include "utils.h"
#include "stopwatch.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void kernel(int *result, 
const double *xcoord, const double *ycoord, int limit, int n)
{
    int t = blockIdx.x*blockDim.x + threadIdx.x;
    if (t < n)
    {
        double x, y, x0, y0;
        x0 = x = xcoord[t];
        y0 = y = ycoord[t];
        for (int i = 0; i < limit; i++)
        {
            if (x * x + y * y >= 4)
            {
                result[t] = i;
                return;
            }
 
            double zx = x * x - y * y + x0;
            y = 2 * x * y + y0;
            x = zx;
        } 
        result[t] = 0;
    }
}

void MandelbrotKernel(int *result, const double *xcoord, const double *ycoord, const int limit, const int n)
{
    size_t dsize = n * sizeof(double);
    size_t isize = n * sizeof(int);
    size_t csize = limit * sizeof(int);
 
    double* d_xcoord;
    hipMalloc(&d_xcoord, dsize);
    double* d_ycoord;
    hipMalloc(&d_ycoord, dsize);
    int* d_result;
    hipMalloc(&d_result, isize);
    int* d_colors;
    hipMalloc(&d_colors, csize);
 
    hipMemcpy(d_xcoord, xcoord, dsize, hipMemcpyHostToDevice);
    hipMemcpy(d_ycoord, ycoord, dsize, hipMemcpyHostToDevice);
    hipMemcpy(d_result, result, isize, hipMemcpyHostToDevice);
 
    int threadsPerBlock = 512;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    kernel<<<blocksPerGrid, threadsPerBlock>>>(d_result, d_xcoord, d_ycoord, limit, n);
 
    hipMemcpy(result, d_result, isize, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    // Free device memory
    hipFree(d_xcoord);
    hipFree(d_ycoord);
    hipFree(d_result);
    hipFree(d_colors);
}
using namespace cimg_library;
using namespace Eigen;

#include <iostream>
__host__
void mandelbrot(ArrayXXd &img, double x0, double y0, double x1, double y1, int max_iter)
{
  int m = img.rows();
  int n = img.cols();
  
  auto x = Eigen::ArrayX<double>::LinSpaced(n,x0,x1);
  auto y = Eigen::ArrayX<double>::LinSpaced(m,y0,y1);

  ArrayXXd X = x.replicate(1,m).transpose();
  ArrayXXd Y = y.replicate(1,m);
  ArrayXXi tmp(m,n);
  MandelbrotKernel(tmp.data(), X.data(), Y.data(), max_iter, m*n);
  img = tmp.cast<double>();
}

int main(int argc, char **argv)
{
  int n = 2048; // image size
  int rep = 1;  // number of repetitions for measurements
  if (argc>1)
    n = std::atoi(argv[1]);
  if (argc>2)
    rep = std::atoi(argv[2]);
  
  ArrayXXd img(n,n);
  StopWatch t(true);
  for(int i=0; i<rep; ++i) {
    mandelbrot(img, 0.273771332381423218946, 0.595859541361479164066, 0.273771332946091993361, 0.595859541784980744876, 10000);
  }
  t.stop();

  std::cout << "Running time: " << double(t.elapsed())/1000/rep << "s";
  if(rep>1)
    std::cout << " (average time for computing a single image)";
  std::cout << "\n";
  std::cout << "Save file to disk...\n";
  save_image("mandelbrot.jpg", img);
  return 0;
}


